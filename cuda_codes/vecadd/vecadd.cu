#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void vecAddKernel(float *A, float *B, float *res, int n) {
	int i = threadIdx.x + blockDim.x * blockIdx.x;
	if (i < n) {
		res[i] = A[i] + B[i];
	}

}

void vecAdd(float *A, float *B, float *res, int n) {
	float *A_d, *B_d, *res_d;
	size_t size = n * sizeof(float);

	printf("Allocating");
	hipMalloc((void **)&A_d, size);
	hipMalloc((void **)&B_d, size);
	hipMalloc((void **)&res_d, size);

	printf(" Done Allocating, now copying");
	hipMemcpy(A_d, A, size, hipMemcpyHostToDevice);
	hipMemcpy(B_d, B, size, hipMemcpyHostToDevice);
	printf(" Done Copying , now setting config params");

	const unsigned int numThreads = 32;
	unsigned int numBlocks = 1;
	printf("Kernel time");
	vecAddKernel<<<numBlocks, numThreads>>>(A_d, B_d, res_d, n);

	// Once the exec is done, we move it back from device to host
	hipMemcpy(res, res_d, size, hipMemcpyDeviceToHost);
	printf("haha done copying results?");

	hipFree(A_d);
	hipFree(B_d);
	hipFree(res_d);
	printf("Done freeing mem from device?");
}

int main() {
	const int n = 16;
	float A[n];
	float B[n];
	float res[n];
	printf("Entered the main fn?");
	for (int i = 0; i < n; i += 1) {
		A[i] = float(i);
		B[i] = A[i] / 1000.0f;
	}

	vecAdd(A, B, res, n);
	printf("Done with vecadd, bitches");
	return 0;
}


